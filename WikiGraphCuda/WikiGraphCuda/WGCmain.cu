#include "hip/hip_runtime.h"
//The following code implemented by Oleg Konings in association with Morgan Hough and Gazzaley lab
//A simple implementation of the Floyd-Warshall all-pairs-shortest path algorithm with path reconstruction. This is indended to be used on directed graphs with no negative cycles
//The Adjacency Matrix is in Row-major format, and is implemented both in CUDA on a Nvidia GTX 680 2GB GPU, and in serial CPU code using an Intel i7-3770 3.9 ghz.
#include <algorithm>
#include <iostream>
#include <sstream>
#include <fstream>
#include <utility>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <vector>
#include <string>
#include <cmath>
#include <map>
#include <hip/hip_runtime.h>
#include <ctime>
#include <cassert>
#include "hip/hip_runtime.h"
#include ""
#define pb push_back 
#define all(c) (c).begin(),(c).end()
#include <Windows.h>
#include <MMSystem.h>
#pragma comment(lib, "winmm.lib")
#define _CRTDBG_MAP_ALLOC
#include <crtdbg.h>//to detect host memory leaks
using namespace std;

#define _DTH hipMemcpyDeviceToHost
#define _HTD hipMemcpyHostToDevice

//these can be altered on user depending on data set and type of operation(random test, read from file etc)
#define BLOCK_SIZE 256
#define RANGE 997
#define RANDOM_GSIZE 700
#define FILE_GSIZE 8298//the number of edges in Wiki-Vote.txt if the file test is run
#define INF (1<<22)
#define DO_TEST_RANDOM 1
#define DO_TEST_FROM_FILE 0

//typedef for vector used in path reconstruction
typedef pair<pair<int,int>,int> Piii;

//forward function declarations
bool InitMMTimer(UINT wTimerRes);
void DestroyMMTimer(UINT wTimerRes, bool init);
void _CPU_Floyd(int *G,int *Gpath,int N);
void _showPath(int start,int end,const vector<Piii> &path,const int *D,const int N);
bool _getPath(int curEdge, int nxtEdge,vector<Piii> &path,const int *D, const int *Dpath,const int N);
void _get_full_paths(const int *D, const int *Dpath, const int N);

//CUDA GPU kernel/functions forward declaration
__global__ void _Wake_GPU(int reps);
__global__ void _GPU_Floyd_kernel(int k, int *G,int *P, int N);
void _GPU_Floyd(int *H_G, int *H_Gpath, const int N);

//other optional utility functions
int _read_from_file(int *G,const int N);
void _generateRandomGraph(int *G, int N, int range, int density);
void _generate_result_file(bool success, unsigned int cpu_time, unsigned int gpu_time, int N);


int main(){
	char ch;
	srand(time(NULL));

	if(DO_TEST_RANDOM){//will use the #define(s) to init a random adjacency Matrix of RANDOM_GSIZE size
		const int NumBytes=RANDOM_GSIZE*RANDOM_GSIZE*sizeof(int);
		//host allocations to create Adjancency matrix and result matrices with path matrices
		int *OrigGraph=(int *)malloc(NumBytes);//will be original Adjancency matrix, will NOT be changed
		int *H_G=(int *)malloc(NumBytes);
		int *H_Gpath=(int *)malloc(NumBytes);
		int *D_G=(int *)malloc(NumBytes);
		int *D_Gpath=(int *)malloc(NumBytes);

		_generateRandomGraph(OrigGraph,RANDOM_GSIZE,RANGE,25);//init graph with values

		cout<<"Successfully created random highly connected graph in adjacency Matrix form with "<<RANDOM_GSIZE*RANDOM_GSIZE<< " elements.\n";
		cout<<"Also created 2 pairs of distinct result Matrices to store the respective results of the CPU results and the GPU results.\n";
		for(int i=0;i<RANDOM_GSIZE*RANDOM_GSIZE;i++){//copy for use in computation
			H_G[i]=D_G[i]=OrigGraph[i];//copy for use in computation
			H_Gpath[i]=D_Gpath[i]=-1;//set to all negative ones for use in path construction
		}
		unsigned int cpu_time=0,gpu_time=0;
		cout<<"\nFloyd-Warshall on CPU underway:\n";
		UINT wTimerRes = 0;
		bool init = InitMMTimer(wTimerRes);
		DWORD startTime = timeGetTime();

		_CPU_Floyd(H_G,H_Gpath,RANDOM_GSIZE);//find shortest paths (with path construction) on serial CPU (Intel i7 3770 3.9 ghz)

		DWORD endTime = timeGetTime();
		cpu_time=unsigned int(endTime-startTime);
		printf("CPU Timing: %dms\n", cpu_time);
		DestroyMMTimer(wTimerRes, init);
		//wake up GPU from idle
		cout<<"\nFloyd-Warshall on GPU underway:\n";
		_Wake_GPU<<<1,BLOCK_SIZE>>>(32);

		//call host function which will copy all info to device and run CUDA kernels
		wTimerRes = 0;
		init = InitMMTimer(wTimerRes);
		startTime = timeGetTime();

		_GPU_Floyd(D_G,D_Gpath,RANDOM_GSIZE);

		endTime = timeGetTime();
		gpu_time=unsigned int(endTime-startTime);
		printf("GPU Timing(including all device-host, host-device copies, device allocations and freeing of device memory): %dms\n\n", gpu_time);
		DestroyMMTimer(wTimerRes, init);

		//compare the device generated result against the host generated result
		cout<<"Verifying results of final adjacency Matrix and Path Matrix.\n";

		int same_adj_Matrix = memcmp(H_G,D_G,NumBytes);
		if(same_adj_Matrix==0){
			cout<<"Adjacency Matrices Equal!\n";
		}else
			cout<<"Adjacency Matrices Not Equal!\n";

		int same_path_Matrix = memcmp(H_Gpath,D_Gpath,NumBytes);
		if(same_path_Matrix==0){
			cout<<"Path reconstruction Matrices Equal!\n";
		}else
			cout<<"Path reconstruction Matrices Not Equal!\n";

		_get_full_paths(D_G,D_Gpath,RANDOM_GSIZE);//find out exact step-by-step shortest paths between vertices(if such a path exists)

		_generate_result_file( bool(same_adj_Matrix==0 && same_path_Matrix==0),cpu_time,gpu_time,RANDOM_GSIZE);

		free(OrigGraph);
		free(H_G);
		free(H_Gpath);
		free(D_G);
		free(D_Gpath);
	}

	_CrtDumpMemoryLeaks();
	cin>>ch;
	return 0;
}

bool InitMMTimer(UINT wTimerRes){
	TIMECAPS tc;
	if (timeGetDevCaps(&tc, sizeof(TIMECAPS)) != TIMERR_NOERROR) {return false;}
	wTimerRes = min(max(tc.wPeriodMin, 1), tc.wPeriodMax);
	timeBeginPeriod(wTimerRes); 
	return true;
}

void DestroyMMTimer(UINT wTimerRes, bool init){
	if(init)
		timeEndPeriod(wTimerRes);
}

void _CPU_Floyd(int *G,int *Gpath,int N){//standard N^3 algo
	for(int k=0;k<N;++k)for(int i=0;i<N;++i)for(int j=0;j<N;++j){
		int curloc=i*N+j,loca=i*N+k,locb=k*N+j;
		if(G[curloc]>(G[loca]+G[locb])){
			G[curloc]=(G[loca]+G[locb]);
			Gpath[curloc]=k;
		}
	}
}

void _showPath(int start,int end,const vector<Piii> &path,const int *D,const int N){
	cout<<"\nHere is the shortest cost path from "<<start<< " to "<<end<<", at a total cost of "<<D[start*N+end]<<".\n";
	for(int i=path.size()-1;i>=0;--i){
		cout<<"From "<<path[i].first.first<<" to "<<path[i].first.second<<" at a cost of "<<path[i].second<<'\n';
	}
	cout<<'\n';
}

bool _getPath(int curEdge, int nxtEdge,vector<Piii> &path,const int *D, const int *Dpath,const int N){
	int curIdx=curEdge*N+nxtEdge;
	if(D[curIdx]>=INF)return false;
	if(Dpath[curIdx]==-1){//end of backwards retracement
		path.push_back(make_pair(make_pair(curEdge,nxtEdge),D[curIdx]));
		return true;
	}else{//record last edge cost and move backwards
		path.push_back(make_pair(make_pair(Dpath[curIdx],nxtEdge),D[Dpath[curIdx]*N+nxtEdge]));
		return _getPath(curEdge,Dpath[curIdx],path,D,Dpath,N);
	}
}

void _get_full_paths(const int *D, const int *Dpath, const int N){
	int start_vertex=-1,end_vertex=-1;
	vector<Piii> path;
	do{
		path.clear();
		cout<<"Enter start vertex #:";
		cin>>start_vertex;
		cout<<"Enter dest vertex(enter negative number to exit) #:";
		cin>>end_vertex;
		if(start_vertex<0 || start_vertex>=N || end_vertex<0 || end_vertex>=N)return;

		if(_getPath(start_vertex, end_vertex,path,D,Dpath,N)){
			_showPath(start_vertex,end_vertex,path,D,N);

		}else{
			cout<<"\nThere does not exist valid a path between "<<start_vertex<<" , and "<<end_vertex<<'\n';

		}
	}while(1);
}

__global__ void _Wake_GPU(int reps){
	int idx=blockIdx.x*blockDim.x + threadIdx.x;
	if(idx>=reps)return;
}

__global__ void _GPU_Floyd_kernel(int k, int *G,int *P, int N){//G will be the adjacency matrix, P will be path matrix
	int col=blockIdx.x*blockDim.x + threadIdx.x;
	if(col>=N)return;
	int idx=N*blockIdx.y+col;

	__shared__ int best;
	if(threadIdx.x==0)
		best=G[N*blockIdx.y+k];
	__syncthreads();
	if(best==INF)return;
	int tmp_b=G[k*N+col];
	if(tmp_b==INF)return;
	int cur=best+tmp_b;
	if(cur<G[idx]){
		G[idx]=cur;
		P[idx]=k;
	}
}
void _GPU_Floyd(int *H_G, int *H_Gpath, const int N){
	//allocate device memory and copy graph data from host
	int *dG,*dP;
	int numBytes=N*N*sizeof(int);
	hipError_t err=hipMalloc((int **)&dG,numBytes);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMalloc((int **)&dP,numBytes);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	//copy from host to device graph info
	err=hipMemcpy(dG,H_G,numBytes,_HTD);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMemcpy(dP,H_Gpath,numBytes,_HTD);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	dim3 dimGrid((N+BLOCK_SIZE-1)/BLOCK_SIZE,N);

	for(int k=0;k<N;k++){//main loop

		_GPU_Floyd_kernel<<<dimGrid,BLOCK_SIZE>>>(k,dG,dP,N);
		err = hipDeviceSynchronize();
		if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	}
	//copy back memory
	err=hipMemcpy(H_G,dG,numBytes,_DTH);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipMemcpy(H_Gpath,dP,numBytes,_DTH);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}

	//free device memory
	err=hipFree(dG);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
	err=hipFree(dP);
	if(err!=hipSuccess){printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);}
}

void _generateRandomGraph(int *G,int N,int range, int density){//density will be between 0 and 100, indication the % of number of directed edges in graph
	//range will be the range of edge weighting of directed edges
	int Prange=(100/density);
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			if(i==j){//set G[i][i]=0
				G[i*N+j]=0;
				continue;
			}
			int pr=rand()%Prange;
			G[i*N+j]= pr==0 ? ((rand()%range)+1):INF;//set edge random edge weight to random value, or to INF
		}
	}
}

int _read_from_file(int *G,const int N){//reads in edge list from file
	int num_edges=0;
	
	ifstream readfile;//enable stream for reading file
	readfile.open("Wiki-Vote.txt");
	assert(readfile.good());//make sure it finds the file & file is
	string line;
	int v0,v1;
	while(getline(readfile,line)){
		istringstream linestream(line);
		linestream>>v0>>v1;
		G[v0*N+v1]=1;
		num_edges++;
	}
	readfile.close();
	return num_edges;
}

void _generate_result_file(bool success, unsigned int cpu_time,unsigned int gpu_time, int N){

	if(!success){
		cout<<"Error in calculation!\n";
		return;
	}else{
		ofstream myfile;
		myfile.open("Floyd-Warshall_result.txt");
		myfile<<"Success! The GPU Floyd-Warshall result and the CPU Floyd-Warshall results are identical(both final adjacency matrix and path matrix).\n\n";
		myfile<<"N= "<<N<<" , and the total number of elements(for Adjacency Matrix and Path Matrix) was "<<N*N<<" .\n";
		myfile<<"Matrices are int full dense format(row major) with a minimum of "<<(N*N)/4<<" valid directed edges.\n\n";
		myfile<<"The CPU timing for all was "<<float(cpu_time)/1000.0f<<" seconds, and the GPU timing(including all device memory operations(allocations,copies etc) ) for all was "<<float(gpu_time)/1000.0f<<" seconds.\n";
		myfile<<"The GPU result was "<<float(cpu_time)/float(gpu_time)<<" faster than the CPU version.\n";
		myfile.close();
	}
}